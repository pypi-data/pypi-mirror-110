#include "hip/hip_runtime.h"
#include <arbor/fvm_types.hpp>
#include <backends/gpu/gpu_common.hpp>

namespace arb {
namespace gpu {

__global__
void multiply_in_place_(fvm_value_type* s, const fvm_index_type* p, int n) {
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    if (tid_<n) {
        s[tid_] *= p[tid_];
    }
}

void multiply_in_place(fvm_value_type* s, const fvm_index_type* p, int n) {
    unsigned block_dim = 128;
    unsigned grid_dim = gpu::impl::block_count(n, block_dim);

    multiply_in_place_<<<grid_dim, block_dim>>>(s, p, n);
}

} // namespace gpu
} // namespace arb
